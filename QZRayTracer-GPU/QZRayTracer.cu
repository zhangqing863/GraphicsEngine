#include "hip/hip_runtime.h"
﻿
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "./src/core/QZRayTracer.h"
#include "./src/core/api.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STB_IMAGE_IMPLEMENTATION
#include "./src/core/stb_image.h"
#include "./src/core/stb_image_write.h"
#include "src/scene/example.h"

using namespace raytracer;
using namespace std;
#define MAXBOUNDTIME 50
#define MAXNUMSHAPE 2000
#define MAXNUMTEXTURE 20

// GPU Mode
// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ Point3f Color(const Ray& r, Shape** world, hiprandState* local_rand_state) {
    Ray cur_ray = r;
    Point3f cur_attenuation = Point3f(1.0f, 1.0f, 1.0f);
    Point3f cur_emitted = Point3f(0.0f, 0.0f, 0.0f);
    for (int i = 0; i < 50; i++) {
        HitRecord rec;

        if ((*world)->Hit(cur_ray, rec)) {
            Ray scattered;
            Point3f attenuation;
            Point3f emitted = rec.mat->Emitted(rec.u, rec.v, rec.p);
            Point3f target = rec.p + Point3f(rec.normal) + RandomInUnitSphere(local_rand_state);
            if (rec.mat->Scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation = cur_attenuation * attenuation + emitted;
                cur_emitted = Point3f(emitted);
                cur_ray = scattered;
            }
            else {
                return cur_attenuation * emitted;
            }
        }
        else {
            /*Vector3f unit_direction = Normalize(cur_ray.d);
            float t = 0.5f * (unit_direction.y + 1.0f);
            Point3f c = Lerp(t, Point3f(1.0, 1.0, 1.0), Point3f(0.5, 0.7, 1.0));*/
            return cur_attenuation * cur_emitted;
        }
    }
        
    return Point3f(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(2022, pixel_index, 0, &rand_state[pixel_index]);
}


__global__ void render(Point3f* fb, int max_x, int max_y, int ns, Camera** cam, Shape** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    Point3f color;
    for (int s = 0; s < ns; s++) {
        Float u = Float(i + hiprand_uniform(&local_rand_state)) / Float(max_x);
        Float v = Float(/*max_y -*/ j /*- 1*/ + hiprand_uniform(&local_rand_state)) / Float(max_y);
        Ray ray = (*cam)->GenerateRay(u, v, &local_rand_state);
        //printf("GetColor。。。\n");
        color += Color(ray, world, &local_rand_state);

        //printf("GetColor done\n");
    }
    rand_state[pixel_index] = local_rand_state;
    color /= Float(ns);
    /*if (Vector3f(color).LengthSquared() > 0) {
        color = Point3f(Normalize(Vector3f(color)));
    }*/
    color = Clamp(color, 0.f, 1.f);
    color = Point3f(pow(color.x, Gamma), pow(color.y, Gamma), pow(color.z, Gamma)); // gamma矫正
    fb[pixel_index] = color;
}


int main() {
    int nx = 2400;
    int ny = 1200;
    int ns = 10000;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    // allocate FB
    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(Point3f);


    size_t size;
    hipDeviceSetLimit(hipLimitMallocHeapSize, 256 * 1024 * 1024);
    hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
    //printf("size: %d\n", size);

    Point3f* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init << <1, 1 >> > (d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables
    Shape** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, MAXNUMSHAPE * sizeof(Shape*)));
    Shape** d_nodes;
    checkCudaErrors(hipMalloc((void**)&d_nodes, MAXNUMSHAPE * sizeof(Shape*)));
    Shape** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Shape*)));
    Camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(Camera*)));



    int image_width, image_height, image_channel;
    unsigned char* image = stbi_load("./resource/texture/signs.jpg", &image_width, &image_height, &image_channel, 0);
    dim3  image_dimensions = dim3(image_width, image_height, image_channel);
    hipExtent volumeSizeBytes = make_hipExtent(sizeof(unsigned char) * image_dimensions.x, image_dimensions.y, image_dimensions.z);
    hipPitchedPtr devicePitchedPointer;
    hipMalloc3D(&devicePitchedPointer, volumeSizeBytes);
    hipMemcpy(devicePitchedPointer.ptr, image, image_width * image_height * image_channel * sizeof(unsigned char), hipMemcpyHostToDevice);


    //printf("Malloc Success\n");
    stbi_image_free(image);
    
    /*--------------------------更换自己的场景--------------------------*/
    Chapter6LightScene2 <<<1, 1>>>(d_list, d_nodes, d_world, d_camera, nx, ny, d_rand_state2, devicePitchedPointer);
    //SampleScene<<<1, 1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    // create_world << <1, 1 >> > (d_list, d_world, d_camera, nx, ny);
    /*------------------------------end--------------------------------*/
    
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render <<<blocks, threads>>> (fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";
    auto* data = (unsigned char*)malloc(nx * ny * 3);
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].x) % 257;
            int ig = int(255.99 * fb[pixel_index].y) % 257;
            int ib = int(255.99 * fb[pixel_index].z) % 257;
            size_t shadingPoint = ((ny - j - 1) * nx + i) * 3;
            data[shadingPoint + 0] = ir;
            data[shadingPoint + 1] = ig;
            data[shadingPoint + 2] = ib;
        }
    }
    // 写入图像
    raytracer::stbi_write_png("./output/RayTracingTheNextWeek/Chapter06-light(cornellbox2).png", nx, ny, 3, data, 0);
    raytracer::stbi_image_free(data);

    // clean up
    checkCudaErrors(hipGetLastError());;
    checkCudaErrors(hipDeviceSynchronize());
    free_world_bvh <<<1, 1 >>> (d_list, d_world, d_camera);
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_nodes));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));
    //checkCudaErrors(hipFree(d_textures));
    //checkCudaErrors(hipFree(devicePitchedPointer));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}

#pragma endregion




